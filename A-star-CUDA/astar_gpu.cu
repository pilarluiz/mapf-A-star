#include "hip/hip_runtime.h"
#include <stdio.h>
#include "astar_gpu.h"
#include "heap.h"
#include "list.h"
#include "sliding_puzzle.h"
#include "pathfinding.h"
#include "cuda_utils.h"
#include "assert.h"
#include <vector>
#include <string>
#include <sstream>
#include <algorithm>
#include <fstream>
#include <chrono>

#define STATES (32 * 1024ll * 1024)
#define HASH_SIZE  (1024 * 1024)
#define HASH_FUNS 128

__global__ void init_heap(const char *s, heap **Q, state *states_pool, char *nodes_pool, int state_len);
__global__ void clear_list(list *S);
__global__ void fill_list(const char *t, int k, int state_len,
		heap **Q, list *S, state *states_pool, char *nodes_pool,
		char ***expand_buf, expand_fun expand, heur_fun h, states_delta_fun states_delta);
__global__ void deduplicate(state **H, list *T, const char *t, heur_fun h);
__global__ void push_to_queues(const char *t, int k, heap **Q, list *S, heur_fun h, int off);

__device__ int f(const state *x, const char *t, heur_fun h);
__device__ int calculate_id();
__device__ state *state_create(const char *node, int f, int g, state *prev,
		state *states_pool, char *nodes_pool, int state_len);

char ***expand_bufs_create(int bufs, int elements, int element_size);
char **expand_buf_create(int elements, int element_size);

void states_pool_create(state **states, char **nodes, int node_size);
void states_pool_destroy(state *states_pool, char *nodes_pool);

#define THREADS_PER_BLOCK  1024
#define BLOCKS 16
#define RESULT_LEN (1024 * 1024)

__device__ int total_Q_size = 0;
__device__ int found = 0;
__device__ int out_of_memory = 0;
__device__ char result_path[RESULT_LEN];

void astar_gpu(const char *s_in, const char *t_in, version_value version, std::fstream &output) {
	char *s_gpu, *t_gpu;
	int k = THREADS_PER_BLOCK * BLOCKS;
	expand_fun expand_fun_cpu;
	heur_fun h_cpu;
	states_delta_fun states_delta_cpu;
	int expand_elements;
	int expand_element_size;

	auto start = std::chrono::high_resolution_clock::now();
	if (version == SLIDING) {
		sliding_puzzle_preprocessing(s_in, t_in, &s_gpu, &t_gpu, &expand_fun_cpu, &h_cpu, &states_delta_cpu,
				&expand_elements, &expand_element_size);
	} else if (version == PATHFINDING) {
		pathfinding_preprocessing(s_in, t_in, &s_gpu, &t_gpu, &expand_fun_cpu, &h_cpu, &states_delta_cpu,
				&expand_elements, &expand_element_size);
	}

	state **H;
	char ***expand_buf = expand_bufs_create(THREADS_PER_BLOCK * BLOCKS, expand_elements, expand_element_size);
	HANDLE_RESULT(hipMalloc(&H, HASH_SIZE * sizeof(state*)));
	HANDLE_RESULT(hipMemset(H, 0, HASH_SIZE * sizeof(state*)));
	heap **Q = heaps_create(k);
	list **Ss = lists_create(BLOCKS, 1000000);
	list *S = list_create(1024 * 1024);
	state *states_pool;
	char *nodes_pool;
	states_pool_create(&states_pool, &nodes_pool, expand_element_size);
	int total_Q_size_cpu;
	int found_cpu;
	int out_of_memory_cpu;

	init_heap<<<1, 1>>>(s_gpu, Q, states_pool, nodes_pool, expand_element_size);
	int step = 0;
	do {
		clear_list<<<1, 1>>>(S);
		HANDLE_RESULT(hipDeviceSynchronize());
		fill_list<<<BLOCKS, THREADS_PER_BLOCK>>>(t_gpu, k, expand_element_size, Q, S, states_pool, nodes_pool,
				expand_buf, expand_fun_cpu, h_cpu, states_delta_cpu);
		HANDLE_RESULT(hipMemcpyFromSymbol(&found_cpu, HIP_SYMBOL(found), sizeof(int)));
		HANDLE_RESULT(hipMemcpyFromSymbol(&out_of_memory, HIP_SYMBOL(found), sizeof(int)));
		if (found_cpu) break;
		if (out_of_memory_cpu) break;
		HANDLE_RESULT(hipDeviceSynchronize());
		deduplicate<<<BLOCKS, THREADS_PER_BLOCK>>>(H, S, t_gpu, h_cpu);
		HANDLE_RESULT(hipDeviceSynchronize());
		push_to_queues<<<1, THREADS_PER_BLOCK>>>(t_gpu, k, Q, S, h_cpu, step) ;
		HANDLE_RESULT(hipDeviceSynchronize());
		HANDLE_RESULT(hipMemcpyFromSymbol(&total_Q_size_cpu, HIP_SYMBOL(total_Q_size), sizeof(int)));
		step++;
	} while (total_Q_size_cpu > 0);

	auto end = std::chrono::high_resolution_clock::now();

	auto duration = end - start;
	output << std::chrono::duration_cast<std::chrono::milliseconds>(duration).count() << "\n";

	if (found_cpu) {
		char result_path_cpu[RESULT_LEN];
		HANDLE_RESULT(hipMemcpyFromSymbol(result_path_cpu, HIP_SYMBOL(result_path), RESULT_LEN));

		std::string path_str = std::string(result_path_cpu);
		std::istringstream path_stream;
		path_stream.str(result_path_cpu);

		std::vector<std::string> v;
		for (std::string line; std::getline(path_stream, line); ) {
			v.push_back(line);
		}
		std::reverse(v.begin(), v.end());
		if (version == SLIDING) {
			output << sliding_puzzle_postprocessing(v);
		} else if (version == PATHFINDING) {
			for (std::string path_el: v) {
				output << path_el << "\n";
			}
		}
	}

	states_pool_destroy(states_pool, nodes_pool);
	lists_destroy(Ss, BLOCKS);
	heaps_destroy(Q, k);
	HANDLE_RESULT(hipFree(H));
	HANDLE_RESULT(hipDeviceSynchronize());
}


__global__ void init_heap(const char *s, heap **Q, state *states_pool, char *nodes_pool, int state_len) {
	heap_insert(Q[0], state_create(s, 0, 0, NULL, states_pool, nodes_pool, state_len));
	atomicAdd(&total_Q_size, 1);
}

__device__ int processed = 0;
__device__ int steps = 0;
__device__ int heaps_min_before;

__global__ void clear_list(list *S) {
	list_clear(S);
}

__global__ void fill_list(const char *t, int k, int state_len,
		heap **Q, list *S, state *states_pool, char *nodes_pool,
		char ***expand_buf, expand_fun expand, heur_fun h, states_delta_fun states_delta) {
	state *m = NULL;
	int id = calculate_id();
	char **my_expand_buf = expand_buf[id];
	if (id == 0)steps++;
	for (int i = id; i < k; i += blockDim.x * gridDim.x) {
		if (Q[i]->size == 0) continue;
		state *q = heap_extract(Q[i]);
		atomicSub(&total_Q_size, 1);
		if (cuda_str_eq(q->node, t)) {
			if (m == NULL || f(q, t, h) < f(m, t, h)) {
				m = q;
			}
			continue;
		}
		expand(q->node, my_expand_buf);
		for (int j = 0; my_expand_buf[j][0] != '\0'; j++) {
			int delta = states_delta(q->node, my_expand_buf[j]);
			state *new_state = state_create(my_expand_buf[j], -1, q->g + delta, q, states_pool, nodes_pool, state_len);
			if (new_state == NULL) return;
			list_insert(S, new_state);
		}
	}
	if (m != NULL && f(m, t, h) <= heaps_min(Q, k)) {
		int found_before = atomicCAS(&found, 0, 1);
		if (found_before == 1) return;

		state *cur = m;
		int result_len = 0;
		while (cur != NULL) {
			int len = cuda_strlen(cur->node) + 1;
			memcpy(result_path + result_len, cur->node, len);
			result_len += len;
			result_path[result_len-1] = '\n';
			cur = cur->prev;
		}
		result_path[result_len-1] = '\0';
		return;
	}
}

__global__ void deduplicate(state **H, list *T, const char *t, heur_fun h) {
	int id = calculate_id();
	for (int i = id; i < T->length; i += blockDim.x * gridDim.x) {
		int z = 0;
		state *t1 = list_get(T, i);
		for (int j = 0; j < HASH_FUNS; j++) {
			assert(t1->node != NULL);
			state *el = H[jenkins_hash(j, t1->node) % HASH_SIZE];
			if (el == NULL || cuda_str_eq(t1->node, el->node)) {
				z = j;
				break;
			}
		}
		int index = jenkins_hash(z, t1->node) % HASH_SIZE;
		t1 = (state*)atomicExch((unsigned long long*)&(H[index]), (unsigned long long)t1);
		if (t1 != NULL && cuda_str_eq(t1->node, list_get(T, i)->node) &&
				f(list_get(T, i), t, h) >= f(t1, t, h)) {
			list_remove(T, i);
			continue;
		}
		t1 = list_get(T, i);
		for (int j = 0; j < HASH_FUNS; j++) {
			if (j != z) {
				state *el = H[jenkins_hash(j, t1->node) % HASH_SIZE];
				if (el != NULL && cuda_str_eq(el->node, t1->node) &&
						f(list_get(T, i), t, h) >= f(el, t, h)) {
					list_remove(T, i);
					break;
				}
			}
		}
	}
}

__global__ void push_to_queues(const char *t, int k, heap **Q, list *S, heur_fun h, int off) {
	for (int i = threadIdx.x; i < S->length; i += blockDim.x) {
		state *t1 = list_get(S, i);
		if (t1 != NULL) {
			t1->f = f(t1, t, h);
			heap_insert(Q[(i + off) % k], t1);
			atomicAdd(&processed, 1);
			atomicAdd(&total_Q_size, 1);
		}
		__syncthreads();
	}
}

__device__ int f(const state *x, const char *t, heur_fun h) {
	return x->g + h(x->node, t);
}

void states_pool_create(state **states, char **nodes, int node_size) {
	HANDLE_RESULT(hipMalloc(states, STATES * sizeof(state)));
	HANDLE_RESULT(hipMalloc(nodes, 3 * STATES * node_size * sizeof(char)));
	HANDLE_RESULT(hipMemset(*states, 0, STATES * sizeof(state)));
	HANDLE_RESULT(hipMemset(*nodes, 0, 3 * STATES * node_size * sizeof(char)));
}

void states_pool_destroy(state *states_pool, char *nodes_pool) {
	HANDLE_RESULT(hipFree(states_pool));
	HANDLE_RESULT(hipFree(nodes_pool));
}

char ***expand_bufs_create(int bufs, int elements, int element_size) {
	int bufs_size = bufs * sizeof(char**);
	char ***bufs_cpu = (char***)malloc(bufs_size);
	for (int i = 0; i < bufs; i++) {
		bufs_cpu[i] = expand_buf_create(elements, element_size);
	}
	char ***bufs_gpu;
	HANDLE_RESULT(hipMalloc(&bufs_gpu, bufs_size));
	HANDLE_RESULT(hipMemcpy(bufs_gpu, bufs_cpu, bufs_size, hipMemcpyDefault));
	free(bufs_cpu);
	return bufs_gpu;

}

char **expand_buf_create(int elements, int element_size) {
	char **buf_cpu = (char**)malloc(elements * sizeof(char*));
	for (int i = 0; i < elements; i++) {
		HANDLE_RESULT(hipMalloc(&(buf_cpu[i]), element_size));
	}
	char **buf_gpu;
	HANDLE_RESULT(hipMalloc(&buf_gpu, elements * sizeof(char*)));
	HANDLE_RESULT(hipMemcpy(buf_gpu, buf_cpu, elements * sizeof(char*),
				hipMemcpyDefault));
	free(buf_cpu);
	return buf_gpu;

}

__device__ int used_states = 0;
__device__ state *state_create(const char *node, int f, int g, state *prev,
		state *states_pool, char *nodes_pool, int state_len) {
	int index = atomicAdd(&used_states, 1);
	if (index >= STATES || (long long)state_len * index >= (1<<30)) {
		out_of_memory = 1;
		return NULL;
	}
	state *result = &(states_pool[index]);
	memcpy(&(nodes_pool[(unsigned long long)state_len * index]), node, state_len);
	result->node = &(nodes_pool[state_len * index]);
	result->f = f;
	result->g = g;
	result->prev = prev;
	return result;
}

__device__ int calculate_id() {
	return threadIdx.x + blockIdx.x * blockDim.x;
}
