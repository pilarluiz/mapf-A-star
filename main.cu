#include "hip/hip_runtime.h"
#include <iostream>
#include <string.h>
#include <fstream>

#include "astar_gpu.h"
#include "sliding_puzzle.h"
#include "pathfinding.h"

struct config {
	version_value version;
	std::string input_file;
	std::string output_file;
};

config parse_args(int argc, const char *argv[]);

char** expand(const char *str) {
	int node = atoi(str);
	char **result = (char**)malloc(4 * sizeof(char*));
	char *arr = (char*)malloc(3 * 11 * sizeof(char*));
	int sibling = node + 1;
	int lchild = 2 * node;
	int rchild = 2 * node + 1;
	sprintf(arr, "%04d %04d %04d ", sibling, lchild, rchild);
	for (int i = 0; i < 3; i++) {
		result[i] = &(arr[5 * i]);
		arr[5 * i - 1] = '\0';
	}
	if (node == 1) {
		result[2] = NULL;
	}
	result[3] = NULL;
	return result;
}

int h(const char *x, const char *t) {
	int res = 0;
	if (atoi(x) > atoi(t)) return 20000;
	int dist = abs(atoi(x) - atoi(t));
	while (dist > 0) {
		dist /= 2;
		res++;
	}
	return res;
}

int map_cpu[10];

#define SLIDING_N  5
#define SLIDING_STATE_LEN (SLIDING_N * SLIDING_N;

int main(int argc, const char *argv[]) {
	config config;
	try {
		config = parse_args(argc, argv);
	} catch (std::string error) {
		std::cout << error << std::endl;
		return 1;
	}
	std::ifstream file(config.input_file);
	std::fstream file_out(config.output_file, std::fstream::out | std::fstream::trunc);
	if (config.version == SLIDING) {
		std::string s, t;
		std::getline(file, s);
		std::getline(file, t);
		astar_gpu(s.c_str(), t.c_str(), SLIDING, file_out);
	} else if (config.version == PATHFINDING) {
		std::string s, t;
		pathfinding_read_input(file, s, t);
		astar_gpu(s.c_str(), t.c_str(), PATHFINDING, file_out);
	}
	return 0;
}

std::string usage(std::string filename) {
	return "Usage: " + filename + " --version [sliding | pathfinding]" +
		" --input-data input.txt --output-data output.txt";
}

config parse_args(int argc, const char *argv[]) {
	config result = {};
	std::string filename = std::string(argv[0]);
	if (argc != 7) throw usage(filename);

	if (std::string(argv[1]) != "--version") throw usage(filename);
	std::string version = std::string(argv[2]);
	if (version == "sliding") result.version = SLIDING;
	else if (version == "pathfinding") result.version = PATHFINDING;
	else throw usage(filename);

	if (std::string(argv[3]) != "--input-data") throw usage(filename);
	result.input_file = std::string(argv[4]);

	if (std::string(argv[5]) != "--output-data") throw usage(filename);
	result.output_file = std::string(argv[6]);
	return result;
}

